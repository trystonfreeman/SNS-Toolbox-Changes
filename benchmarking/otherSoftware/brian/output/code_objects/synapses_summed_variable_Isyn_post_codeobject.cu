#include "hip/hip_runtime.h"
#include "code_objects/synapses_summed_variable_Isyn_post_codeobject.h"
#include "objects.h"
#include "brianlib/common_math.h"
#include "brianlib/cuda_utils.h"
#include "brianlib/stdint_compat.h"
#include <cmath>
#include <stdint.h>
#include <ctime>
#include <stdio.h>



////// SUPPORT CODE ///////
namespace {
    randomNumber_t _host_rand(const int _vectorisation_idx);
    randomNumber_t _host_randn(const int _vectorisation_idx);
    int32_t _host_poisson(double _lambda, const int _vectorisation_idx);

    ///// block extra_device_helper /////

    ///// support_code_lines /////
        
    inline __host__ __device__
    double _brian_clip(const double value,
                              const double a_min,
                              const double a_max)
    {
        if (value < a_min)
            return a_min;
        if (value > a_max)
            return a_max;
        return value;
    }
    template < typename T1, typename T2 > struct _higher_type;
    template < > struct _higher_type<int,int> { typedef int type; };
    template < > struct _higher_type<int,long> { typedef long type; };
    template < > struct _higher_type<int,long long> { typedef long long type; };
    template < > struct _higher_type<int,float> { typedef float type; };
    template < > struct _higher_type<int,double> { typedef double type; };
    template < > struct _higher_type<long,int> { typedef long type; };
    template < > struct _higher_type<long,long> { typedef long type; };
    template < > struct _higher_type<long,long long> { typedef long long type; };
    template < > struct _higher_type<long,float> { typedef float type; };
    template < > struct _higher_type<long,double> { typedef double type; };
    template < > struct _higher_type<long long,int> { typedef long long type; };
    template < > struct _higher_type<long long,long> { typedef long long type; };
    template < > struct _higher_type<long long,long long> { typedef long long type; };
    template < > struct _higher_type<long long,float> { typedef float type; };
    template < > struct _higher_type<long long,double> { typedef double type; };
    template < > struct _higher_type<float,int> { typedef float type; };
    template < > struct _higher_type<float,long> { typedef float type; };
    template < > struct _higher_type<float,long long> { typedef float type; };
    template < > struct _higher_type<float,float> { typedef float type; };
    template < > struct _higher_type<float,double> { typedef double type; };
    template < > struct _higher_type<double,int> { typedef double type; };
    template < > struct _higher_type<double,long> { typedef double type; };
    template < > struct _higher_type<double,long long> { typedef double type; };
    template < > struct _higher_type<double,float> { typedef double type; };
    template < > struct _higher_type<double,double> { typedef double type; };
    template < typename T1, typename T2 >
    __host__ __device__ static inline typename _higher_type<T1,T2>::type
    _brian_mod(T1 x, T2 y)
    {{
        return x-y*floor(1.0*x/y);
    }}
    template < typename T1, typename T2 >
    __host__ __device__ static inline typename _higher_type<T1,T2>::type
    _brian_floordiv(T1 x, T2 y)
    {{
        return floor(1.0*x/y);
    }}
    #ifdef _MSC_VER
    #define _brian_pow(x, y) (pow((double)(x), (y)))
    #else
    #define _brian_pow(x, y) (pow((x), (y)))
    #endif
                    inline __device__ int _brian_atomicAdd(int* address, int val)
                    {
            // hardware implementation
            return atomicAdd(address, val);
                    }
                    inline __device__ float _brian_atomicAdd(float* address, float val)
                    {
            // hardware implementation
            return atomicAdd(address, val);
                    }
                    inline __device__ double _brian_atomicAdd(double* address, double val)
                    {
                            #if (__CUDA_ARCH__ >= 600)
            // hardware implementation
            return atomicAdd(address, val);
                            #else
            // software implementation
            unsigned long long int* address_as_int = (unsigned long long int*)address;
            unsigned long long int old = *address_as_int, assumed;
            do {
                assumed = old;
                old = atomicCAS(address_as_int, assumed,
                                __double_as_longlong(val +
                                       __longlong_as_double(assumed)));
            // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
            } while (assumed != old);
            return __longlong_as_double(old);
                            #endif
                    }
                    inline __device__ int _brian_atomicMul(int* address, int val)
                    {
                        // software implementation
                        int old = *address, assumed;
                        do {
                            assumed = old;
                            old = atomicCAS(address, assumed, val * assumed);
                        } while (assumed != old);
                        return old;
                    }
                    inline __device__ float _brian_atomicMul(float* address, float val)
                    {
            // software implementation
            int* address_as_int = (int*)address;
            int old = *address_as_int, assumed;
            do {
                assumed = old;
                old = atomicCAS(address_as_int, assumed,
                                __float_as_int(val *
                                       __int_as_float(assumed)));
            // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
            } while (assumed != old);
            return __int_as_float(old);
                    }
                    inline __device__ double _brian_atomicMul(double* address, double val)
                    {
            // software implementation
            unsigned long long int* address_as_int = (unsigned long long int*)address;
            unsigned long long int old = *address_as_int, assumed;
            do {
                assumed = old;
                old = atomicCAS(address_as_int, assumed,
                                __double_as_longlong(val *
                                       __longlong_as_double(assumed)));
            // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
            } while (assumed != old);
            return __longlong_as_double(old);
                    }
                    inline __device__ int _brian_atomicDiv(int* address, int val)
                    {
                        // software implementation
                        int old = *address, assumed;
                        do {
                            assumed = old;
                            old = atomicCAS(address, assumed, val / assumed);
                        } while (assumed != old);
                        return old;
                    }
                    inline __device__ float _brian_atomicDiv(float* address, float val)
                    {
            // software implementation
            int* address_as_int = (int*)address;
            int old = *address_as_int, assumed;
            do {
                assumed = old;
                old = atomicCAS(address_as_int, assumed,
                                __float_as_int(val /
                                       __int_as_float(assumed)));
            // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
            } while (assumed != old);
            return __int_as_float(old);
                    }
                    inline __device__ double _brian_atomicDiv(double* address, double val)
                    {
            // software implementation
            unsigned long long int* address_as_int = (unsigned long long int*)address;
            unsigned long long int old = *address_as_int, assumed;
            do {
                assumed = old;
                old = atomicCAS(address_as_int, assumed,
                                __double_as_longlong(val /
                                       __longlong_as_double(assumed)));
            // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
            } while (assumed != old);
            return __longlong_as_double(old);
                    }


    // Implement dummy functions such that the host compiled code of binomial
    // functions works. Hacky, hacky ...
    randomNumber_t _host_rand(const int _vectorisation_idx)
    {
        printf("ERROR: Called dummy function `_host_rand` in %s:%d\n", __FILE__,
                __LINE__);
        exit(EXIT_FAILURE);
    }
    randomNumber_t _host_randn(const int _vectorisation_idx)
    {
        printf("ERROR: Called dummy function `_host_rand` in %s:%d\n", __FILE__,
                __LINE__);
        exit(EXIT_FAILURE);
    }
    int32_t _host_poisson(double _lambda, const int _vectorisation_idx)
    {
        printf("ERROR: Called dummy function `_host_poisson` in %s:%d\n", __FILE__,
                __LINE__);
        exit(EXIT_FAILURE);
    }
}

////// hashdefine_lines ///////



__global__ void
_run_kernel_synapses_summed_variable_Isyn_post_codeobject(
    int _N,
    int THREADS_PER_BLOCK,
    ///// KERNEL_PARAMETERS /////
    double* _ptr_array_neurongroup_Isyn,
	int32_t* _ptr_array_synapses_N,
	int32_t* _ptr_array_synapses__synaptic_post,
	const int _num_postsynaptic_idx,
	int32_t* _ptr_array_synapses__synaptic_pre,
	const int _num_presynaptic_idx,
	const int _num_synaptic_post,
	double* _ptr_array_neurongroup_v
    )
{
    using namespace brian;

    int tid = threadIdx.x;
    int bid = blockIdx.x;

    int _idx = bid * THREADS_PER_BLOCK + tid;
    int _vectorisation_idx = _idx;

    ///// KERNEL_CONSTANTS /////
    const int _numIsyn_post = 10;
	const int _numN = 1;
	const int _numv_post = 10;
	const int _numv_pre = 10;

    ///// kernel_lines /////
        


    assert(THREADS_PER_BLOCK == blockDim.x);


    if(_vectorisation_idx >= _N)
    {
        return;
    }



    ///// scalar_code /////
        
    const double _lio_1 = 1.0f*1.0/20.0;


    {
        ///// vector_code /////
                
        const int32_t _postsynaptic_idx = _ptr_array_synapses__synaptic_post[_idx];
        const int32_t _presynaptic_idx = _ptr_array_synapses__synaptic_pre[_idx];
        const double v_post = _ptr_array_neurongroup_v[_postsynaptic_idx];
        const double v_pre = _ptr_array_neurongroup_v[_presynaptic_idx];
        const double _synaptic_var = 0.5 * (_brian_clip(_lio_1 * v_pre, 0, 1) * ((- 60.0) - v_post));


int _target_id = _ptr_array_synapses__synaptic_post[_idx];
_brian_atomicAdd(&_ptr_array_neurongroup_Isyn[_target_id], _synaptic_var);
    }
}


void _run_synapses_summed_variable_Isyn_post_codeobject()
{
    using namespace brian;


    const int _N = _array_synapses_N[0];

    ///// HOST_CONSTANTS ///////////
    const int _numIsyn_post = 10;
		const int _numN = 1;
		int32_t* const dev_array_synapses__synaptic_post = thrust::raw_pointer_cast(&dev_dynamic_array_synapses__synaptic_post[0]);
		const int _num_postsynaptic_idx = dev_dynamic_array_synapses__synaptic_post.size();
		int32_t* const dev_array_synapses__synaptic_pre = thrust::raw_pointer_cast(&dev_dynamic_array_synapses__synaptic_pre[0]);
		const int _num_presynaptic_idx = dev_dynamic_array_synapses__synaptic_pre.size();
		const int _num_synaptic_post = dev_dynamic_array_synapses__synaptic_post.size();
		const int _numv_post = 10;
		const int _numv_pre = 10;


    static int num_threads, num_blocks;
    static size_t needed_shared_memory = 0;
    static bool first_run = true;
    if (first_run)
    {
        // get number of blocks and threads
        int min_num_threads; // The minimum grid size needed to achieve the
                             // maximum occupancy for a full device launch

        CUDA_SAFE_CALL(
                hipOccupancyMaxPotentialBlockSize(&min_num_threads, &num_threads,
                    _run_kernel_synapses_summed_variable_Isyn_post_codeobject, 0, 0)  // last args: dynamicSMemSize, blockSizeLimit
                );

        // Round up according to array size
        num_blocks = (_N + num_threads - 1) / num_threads;





        // calculate theoretical occupancy
        int max_active_blocks;
        CUDA_SAFE_CALL(
                hipOccupancyMaxActiveBlocksPerMultiprocessor(&max_active_blocks,
                    _run_kernel_synapses_summed_variable_Isyn_post_codeobject, num_threads, needed_shared_memory)
                );

        float occupancy = (max_active_blocks * num_threads / num_threads_per_warp) /
                          (float)(max_threads_per_sm / num_threads_per_warp);


        // check if we have enough ressources to call kernel with given number
        // of blocks and threads (can only occur for the else case above as for the
        // first max. occupancy)
        struct hipFuncAttributes funcAttrib;
        CUDA_SAFE_CALL(
                hipFuncGetAttributes(&funcAttrib, reinterpret_cast<const void*>(_run_kernel_synapses_summed_variable_Isyn_post_codeobject))
                );
        if (num_threads > funcAttrib.maxThreadsPerBlock)
        {
            // use the max num_threads before launch failure
            num_threads = funcAttrib.maxThreadsPerBlock;
            printf("WARNING Not enough ressources available to call "
                   "_run_kernel_synapses_summed_variable_Isyn_post_codeobject "
                   "with maximum possible threads per block (%u). "
                   "Reducing num_threads to %u. (Kernel needs %i "
                   "registers per block, %i bytes of "
                   "statically-allocated shared memory per block, %i "
                   "bytes of local memory per thread and a total of %i "
                   "bytes of user-allocated constant memory)\n",
                   max_threads_per_block, num_threads, funcAttrib.numRegs,
                   funcAttrib.sharedSizeBytes, funcAttrib.localSizeBytes,
                   funcAttrib.constSizeBytes);

            // calculate theoretical occupancy for new num_threads
            CUDA_SAFE_CALL(
                    hipOccupancyMaxActiveBlocksPerMultiprocessor(&max_active_blocks,
                        _run_kernel_synapses_summed_variable_Isyn_post_codeobject, num_threads, needed_shared_memory)
                    );

            occupancy = (max_active_blocks * num_threads / num_threads_per_warp) /
                        (float)(max_threads_per_sm / num_threads_per_warp);
        }


        else
        {
            printf("INFO _run_kernel_synapses_summed_variable_Isyn_post_codeobject\n"
                   "\t%u blocks\n"
                   "\t%u threads\n"
                   "\t%i registers per thread\n"
                   "\t%i bytes statically-allocated shared memory per block\n"
                   "\t%i bytes local memory per thread\n"
                   "\t%i bytes user-allocated constant memory\n"
                   "\t%.3f theoretical occupancy\n",
                   num_blocks,
                   num_threads, funcAttrib.numRegs,
                   funcAttrib.sharedSizeBytes, funcAttrib.localSizeBytes,
                   funcAttrib.constSizeBytes, occupancy);
        }
        first_run = false;
    }

const int _target_size = 10;

// Reset summed variables to zero
CUDA_SAFE_CALL(
        hipMemset(dev_array_neurongroup_Isyn + 0,
                   0,
                   _target_size * sizeof(double))
        );

    _run_kernel_synapses_summed_variable_Isyn_post_codeobject<<<num_blocks, num_threads>>>(
            _N,
            num_threads,
            ///// HOST_PARAMETERS /////
            dev_array_neurongroup_Isyn,
			dev_array_synapses_N,
			dev_array_synapses__synaptic_post,
			_num_postsynaptic_idx,
			dev_array_synapses__synaptic_pre,
			_num_presynaptic_idx,
			_num_synaptic_post,
			dev_array_neurongroup_v
        );

    CUDA_CHECK_ERROR("_run_kernel_synapses_summed_variable_Isyn_post_codeobject");


}


